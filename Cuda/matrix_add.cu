#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <sys/timeb.h>
#include <>
#include <cmath>

__global__ void matrix_add(int *d_A, int *d_B, int *d_OUT, int num_col, int num_row) {
	int col = threadIdx.x + blockDim.x * blockIdx.x;
	int row = threadIdx.y + blockDim.y * blockIdx.y;
	int index = row * num_col + col;
	if (col < num_col && row < num_row) {
		d_OUT[index] = d_A[index] + d_B[index];
	}
}

void matrix_add(int num_col, int num_row) {
	int* A, * B, * OUT;
	int* d_A, * d_B, * d_OUT;
	int size = num_col * num_row;
	A = (int *) malloc(size * sizeof(int));
	B = (int *) malloc(size * sizeof(int));
	OUT = (int *) malloc(size * sizeof(int));

	for (int i = 0; i < size; i++) {
		A[i] = rand() % 100;
		B[i] = rand() % 100;
	}

	hipError_t cudaStatus = hipMalloc((void**)&d_A, sizeof(int) * size);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "A malloc failed");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&d_B, sizeof(int) * size);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "B malloc failed");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&d_OUT, sizeof(int) * size);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "OUT malloc failed");
		goto Error;
	}

	cudaStatus = hipMemcpy(d_A, A, sizeof(int) * size, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "A copy failed");
		goto Error;
	}

	cudaStatus = hipMemcpy(d_B, B, sizeof(int) * size, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "B copy failed");
		goto Error;
	}
	

	dim3 grid(ceil(num_col / 32), ceil(num_row / 32));
	dim3 block(32, 32);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);

	matrix_add<<<grid, block>>>(d_A, d_B, d_OUT, num_col, num_row);

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	
	float time;
	hipEventElapsedTime(&time, start, stop);
	printf("Time: %f\n", time);
	
	hipEventDestroy(start);
	hipEventDestroy(stop);

	cudaStatus = hipMemcpy(OUT, d_OUT, sizeof(int) * size, hipMemcpyHostToDevice);


Error:
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_OUT);
	free(A);
	free(B);
	free(OUT);
}